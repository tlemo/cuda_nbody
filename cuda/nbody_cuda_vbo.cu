#include "hip/hip_runtime.h"

// Structure-of-Arrays + VBO (Vertex Buffer Object)

#include <common/cuda_utils.h>
#include <common/math_2d.h>
#include <common/nbody_plugin.h>
#include <common/utils.h>

#include <GL/glew.h>
#include <GL/gl.h>

#include <cuda_gl_interop.h>

#include <utility>
#include <vector>
#include <math.h>

namespace cuda_vbo {

struct KernelArgs {
  Vector2* prev_pos = nullptr;
  Vector2* pos = nullptr;
  Vector2* vel = nullptr;
  Scalar* mass = nullptr;
  int count = 0;
};

__global__ static void UpdateKernel(KernelArgs args) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < args.count) {
    const auto pos = args.prev_pos[index];
    Vector2 acc = { 0, 0 };
    for (int j = 0; j < args.count; ++j) {
      const Vector2 r = args.prev_pos[j] - pos;
      const Scalar dist_squared = length_squared(r) + kSofteningFactor;
      const Scalar inv_dist = 1 / sqrt(dist_squared);
      const Scalar inv_dist_cube = inv_dist * inv_dist * inv_dist;
      const Scalar s = args.mass[j] * inv_dist_cube;
      acc = acc + r * s;
    }
    const auto v = (args.vel[index] + acc * kTimeStep) * kDampingFactor;
    args.pos[index] = pos + v * kTimeStep;
    args.vel[index] = v;
  }
}

class NBody : public NBodyPlugin {
 public:
  NBody() : NBodyPlugin("cuda_vbo") {}

 private:
  void Free() {
    if (prev_pos_cu_ != nullptr) {
      CHECK(pos_cu_ != nullptr);
      CU(hipGraphicsUnregisterResource(prev_pos_cu_));
      CU(hipGraphicsUnregisterResource(pos_cu_));
      prev_pos_cu_ = nullptr;
      pos_cu_ = nullptr;
    }

    if (prev_pos_vbo_ != 0) {
      CHECK(pos_vbo_ != 0);
      glDeleteBuffers(1, &prev_pos_vbo_);
      glDeleteBuffers(1, &pos_vbo_);
      prev_pos_vbo_ = 0;
      pos_vbo_ = 0;
    }

    CU(hipFree(vel_));
    CU(hipFree(mass_));
    vel_ = nullptr;
    mass_ = nullptr;

    bodies_count_ = 0;
  }

  void Init(const std::vector<Body>& bodies, int, int) final {
    PrintCudaInfo();

    // reset state
    Free();

    bodies_count_ = bodies.size();
    CHECK(bodies_count_ > 0);

    const size_t buffer_size = bodies_count_ * sizeof(Vector2);

    // CUDA buffers
    CU(hipMallocManaged(&vel_, buffer_size));
    CU(hipMallocManaged(&mass_, bodies_count_ * sizeof(Scalar)));

    // OpenGL buffers
    GLint actual_size = 0;

    glGenBuffers(1, &prev_pos_vbo_);
    glBindBuffer(GL_ARRAY_BUFFER, prev_pos_vbo_);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);

    // sanity check
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, &actual_size);
    CHECK(actual_size == buffer_size);

    glGenBuffers(1, &pos_vbo_);
    glBindBuffer(GL_ARRAY_BUFFER, pos_vbo_);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);

    // sanity check
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, &actual_size);
    CHECK(actual_size == buffer_size);

    // finally, setup GL/CUDA interop
    // (this doesn't seem to work after glMapBuffer() was used?)
    // glBindBuffer(GL_ARRAY_BUFFER, 0);
    CU(hipGraphicsGLRegisterBuffer(
        &prev_pos_cu_, prev_pos_vbo_, cudaGraphicsMapFlagsNone));
    CU(hipGraphicsGLRegisterBuffer(
        &pos_cu_, pos_vbo_, cudaGraphicsMapFlagsNone));

    glBindBuffer(GL_ARRAY_BUFFER, pos_vbo_);
    auto pos =
        static_cast<Vector2*>(glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY));
    CHECK(pos != nullptr);

    // copy initial values
    for (int i = 0; i < bodies_count_; ++i) {
      pos[i] = bodies[i].pos;
      vel_[i] = bodies[i].v;
      mass_[i] = bodies[i].mass;
    }

    CHECK(glUnmapBuffer(GL_ARRAY_BUFFER));
    glBindBuffer(GL_ARRAY_BUFFER, 0);
  }

  void Shutdown() final { Free(); }

  void Render() final {
    glColor3f(0.5, 1.0, 0.9);
    glPointSize(1.0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glBindBuffer(GL_ARRAY_BUFFER, pos_vbo_);

    glVertexPointer(2, GL_FLOAT, 0, 0);

    glDrawArrays(GL_POINTS, 0, bodies_count_);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glDisableClientState(GL_VERTEX_ARRAY);
  }

  void Update() final {
    std::swap(pos_vbo_, prev_pos_vbo_);
    std::swap(pos_cu_, prev_pos_cu_);

    // mapping hints
    CU(cudaGraphicsResourceSetMapFlags(pos_cu_,
                                       cudaGraphicsMapFlagsWriteDiscard));
    CU(cudaGraphicsResourceSetMapFlags(prev_pos_cu_,
                                       cudaGraphicsMapFlagsReadOnly));

    // map the resources
    CU(hipGraphicsMapResources(1, &pos_cu_, 0));
    CU(hipGraphicsMapResources(1, &prev_pos_cu_, 0));

    KernelArgs args;
    args.vel = vel_;
    args.mass = mass_;
    args.count = bodies_count_;

    size_t unused = 0;
    CU(hipGraphicsResourceGetMappedPointer(
        reinterpret_cast<void**>(&args.pos), &unused, pos_cu_));
    CU(hipGraphicsResourceGetMappedPointer(
        reinterpret_cast<void**>(&args.prev_pos), &unused, prev_pos_cu_));

    const int kBlockSize = 128;
    const int kBlockCount = (bodies_count_ + kBlockSize - 1) / kBlockSize;
    UpdateKernel<<<kBlockCount, kBlockSize>>>(args);
    CU(hipGetLastError());

    CU(hipGraphicsUnmapResources(1, &prev_pos_cu_, 0));
    CU(hipGraphicsUnmapResources(1, &pos_cu_, 0));
  }

 private:
  GLuint prev_pos_vbo_ = 0;
  GLuint pos_vbo_ = 0;

  hipGraphicsResource* prev_pos_cu_ = nullptr;
  hipGraphicsResource* pos_cu_ = nullptr;

  Vector2* vel_ = nullptr;
  Scalar* mass_ = nullptr;

  int bodies_count_ = 0;
};

static NBody instance;

}  // namespace cuda_vbo
